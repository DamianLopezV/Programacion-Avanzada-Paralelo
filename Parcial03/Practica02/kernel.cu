#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

const int N = 1024;
__global__ void prefixSums(int *a, int*b)
{
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = 0; i < N; i++) {
        if (gid + i < N) {
            b[gid + i] += a[gid];
        }
        __syncthreads();
    }
}

__global__ void prefixSums2(int* a, int* b)
{
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    b[gid] = a[gid];
    for (int i = 1; i < N; i*=2) {
        if (gid + i < N) {
            if ((gid + 1) % i == 0) {
                b[gid + i] += b[gid + i / 2];
            }
        }
        __syncthreads();
    }
}

int main()
{
    int a[N];
    int b[N];
    int* d_a, * d_b;
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));

    for (int i = 0; i < N; i++) {
        a[i] = 1;
        b[i] = 0;
        printf("%d ", a[i]);
    }
    printf("\n\n");

    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    
    prefixSums << <1, N >> > (d_a, d_b);

    hipMemcpy(b, d_b, N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        printf("%d ", b[i]);
    }
    printf("\n\n");

    return 0;
}