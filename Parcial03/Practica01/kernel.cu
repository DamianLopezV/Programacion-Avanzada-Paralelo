#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

const int size = 1024;
const int stream_Size = 8;
__global__ void addKernel(int *a, int *b, int *c)
{
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid < size) {
        c[gid] = a[gid] + b[gid];
    }
}

int main()
{
    int a[size], b[size], c[size];
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    int* d_a, * d_b, * d_c;
    hipMalloc((void**)&d_a, sizeof(int) * size);
    hipMalloc((void**)&d_b, sizeof(int) * size);
    hipMalloc((void**)&d_c, sizeof(int) * size);

    dim3 block(128);
    dim3 grid(size / block.x);
    hipStream_t str[stream_Size];
    for (int i = 0; i < stream_Size; i++) {
        hipStreamCreate(&str[i]);
    }

    for (int i = 0; i < stream_Size; i++) {
        hipMemcpyAsync(d_a, a, sizeof(int) * size, hipMemcpyHostToDevice, str[i]);
        hipMemcpyAsync(d_b, b, sizeof(int) * size, hipMemcpyHostToDevice, str[i]);
        addKernel << <grid, block, 0, str[i] >> > (d_a, d_b, d_c);
        hipMemcpyAsync(c, d_c, sizeof(int) * size, hipMemcpyDeviceToHost, str[i]);
    }

    for (int i = 0; i < size; i++) {
        printf("%d: %d\n", i, c[i]);
    }
    

    return 0;
}


