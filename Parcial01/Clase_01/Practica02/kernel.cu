#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void multiplicationKernel(int* d_a, int* d_b, int* d_c)
{
    d_c[threadIdx.x] = d_a[threadIdx.x] * d_b[threadIdx.x];
}

int main()
{
    const int N = 3;
    int a[N] = { 1,0,1 };
    int b[N] = { 2,4,3 };
    int c[N] = { 0 };
    int size = N * sizeof(int);
    int* d_a, * d_b, * d_c;

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    multiplicationKernel << <1, 3 >> > (d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printf("[1,0,1]*[2,4,3] = [%d,%d,%d]", c[0], c[1], c[2]);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

